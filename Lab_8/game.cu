#include<stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
__global__ void compute(int* x,int* y,int n){
        int col=threadIdx.x+blockIdx.x*blockDim.x;
        int row=threadIdx.y+blockIdx.y*blockDim.y;
        int num=col+row*n;
        int neighbor=0;
        //cell in the middle has eight neighbors,
        //a cell in a corner has only three neighbors,
        //a cell on an edge has five neighbors. 
    if(col<n && row<n){     
        //corner 3
            //In order to move to corner,
            //it should move either diagonal or move left/right and move up/down
        //top left corner
        if(col==0 && row==0){
            neighbor+=x[num+1]; //move right
            neighbor+=x[num+n];  //move bottom
            neighbor+=x[num+n+1]; //bottom right
        }
        //bottom left corner
        else if(col==0 && row==n-1){
            neighbor+=x[num+1];   //move right
            neighbor+=x[num-n];   //move up
            neighbor+=x[num-n+1]; //top right
        }
        //bottom right
        else if(col==n-1 && row==n-1){  
                neighbor+=x[num-1];   //move left
                neighbor+=x[num-n];  //move up
                neighbor+=x[num-n-1]; //top left
        }
        //edge 5
        //In order to move to edge
        // it should just move to left/right/up/down (including corner)
        //top edge
        else if(row==0 && col>0 && col<n-1){    
                neighbor+=x[num-1]; //left
                neighbor+=x[num+1]; //right
                neighbor+=x[num+n-1]; //bottom left -- corner
                neighbor+=x[num+n];   //bottom
                neighbor+=x[num+n+1]; //bottom right -- corner
        }
        //bottom edge
        else if(row==n-1 && col>0 && col<n-1){  
                neighbor+=x[num-1]; //left
                neighbor+=x[num+1]; //right
                neighbor+=x[num-n+1]; //Top right-- corner
                neighbor+=x[num-n];   //Top
                neighbor+=x[num-n-1]; //Top left -- corner   
        }
        //Left edge
        else if(col==0 && row>0 && row<n-1){    
                neighbor+=x[num+1]; //right
                neighbor+=x[num-n]; //Top
                neighbor+=x[num-n+1]; //Top right-- corner
                neighbor+=x[num+n];   //Bottom
                neighbor+=x[num+n+1]; //Bottom right-- corner     
        }
        //Right edge
        else if(col==n-1 && row>0 && row<n-1){  
                neighbor+=x[num-1]; //left
                neighbor+=x[num-n]; //Top
                neighbor+=x[num-n-1]; //Top left-- corner
                neighbor+=x[num+n];   //Bottom
                neighbor+=x[num+n-1]; //Bottom left-- corner       
        }
         //Right edge
        else if(col==n-1 && row>0 && row<n-1){  
                neighbor+=x[num-1]; //left
                neighbor+=x[num-n]; //Top
                neighbor+=x[num-n-1]; //Top left-- corner
                neighbor+=x[num+n];   //Bottom
                neighbor+=x[num+n-1]; //Bottom left-- corner       
        }

        //cell in the middle has eight neighbors,        
        else{   
                neighbor+=x[num-1];//left  
                neighbor+=x[num+1];   //right
                neighbor+=x[num-n-1];//top left
                neighbor+=x[num-n];    //top
                neighbor+=x[num-n+1];  //top right
                neighbor+=x[num+n-1];  //bottom left
                neighbor+=x[num+n];    //bottom
                neighbor+=x[num+n+1];  //bottom right
        }
        //Die : 0
        //Live: 1
        //A live cell with zero or one live neighbor dies from loneliness.
        if(x[num]==1 && (neighbor ==0 || neighbor ==1))
                y[num]=0;
        //A live cell with four or more live neighbors dies due to overpopulation.
        else if(x[num]==1 && neighbor>=4)
                y[num]=0;
        //A dead cell with two or three live neighbors becomes alive.
        else if(x[num]==1 && (neighbor==2 || neighbor==3))
                y[num]=1;
        //Otherwise, a cell's state stays unchanged
        else
            y[num] = x[num];
    }
}


int main(void){
        int i,j,k;
        int row= 6;
        int col= 4;
        int start[row][col];
        int Round[row][col];
        dim3 threadsPerBlock(32,32);
        dim3 numBlocks(row/threadsPerBlock.x,col/threadsPerBlock.x);
        int* x;
        int* y;
        int generation =1;// maximum generation/iteration
        float milliseconds=0;
        hipEvent_t t_start,t_stop;
        hipEventCreate(&t_start);
        hipEventCreate(&t_stop);
        //Initilazie the matrix of the x Generated cell
        for(i=0;i<row;i++)
                for(j=0;j<col;j++)
                        start[i][j]=rand()%2;
         //Initilazie the matrix of the y Generated cell
        for(i=0;i<row;i++)
                for(j=0;j<col;j++)
                        Round[i][j]=0;
   
        hipMalloc((void **) &x,sizeof(int)*row*col);
        hipMemcpy(x,start,sizeof(int)*row*col,hipMemcpyHostToDevice);


        printf("Start\n");
        printf("-------\n");
        for(i=0;i<row;i++)
        {
                for(j=0;j<col;j++)
                {
                    if(start[i][j]){
                        printf(" 0");
                    }
                    else{
                        printf(" 1");
                    }
                }
                printf("\n");
        }

        for(k=0;k<= generation;k++)
        {  
            hipEventRecord(t_start);
            compute<<<numBlocks,threadsPerBlock>>>(y,y,row);
            hipEventRecord(t_stop);
            hipMalloc((void **) &y,sizeof(int)*row*col);

            hipMemcpy(Round,y,sizeof(int)*row*col,hipMemcpyDeviceToHost);

                printf("\n Round %d \n",k);
                printf("-------\n");
                    for(i=0;i<row;i++)
                    {
                        for(j=0;j<col;j++)
                        {
                            if(Round[i][j])
                            {
                                   printf(" 0");
                            }
                            else{
                                printf(" 1");
                            }
                        }
                        printf("\n");
                    }   
        hipEventElapsedTime(&milliseconds,t_start,t_stop);
        printf("Time taken for this computation = %f milliseconds\n\n",milliseconds);
      
    }
        

        return 0;
}


