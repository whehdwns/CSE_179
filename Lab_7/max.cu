
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <stdlib.h>
#include <ctime>
#define N 10000


__global__ void findmaximum(float *A, float *max,int n)
{
  int index = threadIdx.x + blockIdx.x*blockDim.x;
  int dim = gridDim.x*blockDim.x;
  int offset =0;
  float temp;
  while(index + offset < n){
    temp = fmaxf(temp, A[index + offset]);
    offset += dim;
  }
  __syncthreads();

}


int main()
{
  float *h_A;
  float *d_array;
  float *h_max;
  float *d_max;

  // allocate memory
  h_A = (float*)malloc(N*sizeof(float));
  h_max = (float*)malloc(sizeof(float));
  hipMalloc((void**)&d_array, N*sizeof(float));
  hipMalloc((void**)&d_max, sizeof(float));
  hipMemset(d_max, 0, sizeof(float));

  for(int i=0;i<N;i++){
    h_A[i] = N*float(rand()) / RAND_MAX;
  }
  //GPU
  float gpu_elapsed_time;
  hipEvent_t gpu_start;
  hipEvent_t gpu_stop;
  hipEventCreate(&gpu_start);
  hipEventCreate(&gpu_stop);
  //start Time
  hipEventRecord(gpu_start, 0);
  hipMemcpy(d_array, h_A, N*sizeof(float), hipMemcpyHostToDevice);

  for(int j=0;j<1000;j++){
    dim3 gridSize = 128;
    dim3 blockSize = 128;
    findmaximum<<< gridSize, blockSize >>>(d_array, d_max, N);
  }

  hipMemcpy(h_max, d_max, sizeof(float), hipMemcpyDeviceToHost);
  hipEventRecord(gpu_stop, 0);
  hipEventSynchronize(gpu_stop);
  hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
  hipEventDestroy(gpu_start);
  hipEventDestroy(gpu_stop);
  //stop Time
  std::cout<<"Maximum number found on gpu was: "<<*h_max<<std::endl;
  std::cout<<"The gpu took: "<<gpu_elapsed_time<<" milli-seconds"<<std::endl;
  //CPU
  //start time
  clock_t cpu_start = clock();
  for(unsigned int j=0;j<1000;j++){
    *h_max = -1.0;
    for(unsigned int i=0;i<N;i++){
      if(h_A[i] > *h_max){
        *h_max = h_A[i];
      }
    }
  }
  //stop time
  clock_t cpu_stop = clock();
  clock_t cpu_elapsed_time = 1000*(cpu_stop - cpu_start)/CLOCKS_PER_SEC;

  std::cout<<"Maximum number found on cpu was: "<<*h_max<<std::endl;
  std::cout<<"The cpu took: "<<cpu_elapsed_time<<" milli-seconds"<<std::endl;

  free(h_A);
  free(h_max);
  hipFree(d_array);
  hipFree(d_max);
}


